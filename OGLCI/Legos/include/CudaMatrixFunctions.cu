#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void multiplyKernel(int resultWidth, int resultHeight, int leftWidth, int valueColsPerThread, int valueRowsPerThread, double* left, double* right, double* result)
{
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    int totalGridHeight = blockDim.y * gridDim.y * valueRowsPerThread;
    int totalGridWidth = blockDim.x * gridDim.x * valueColsPerThread;

    for (int i = 0; i < valueColsPerThread; ++i)
    {
        int currentRow = ROW + i;
        for (int j = 0; j < valueRowsPerThread; ++j)
        {
            int currentCol = COL + j;
            if (currentRow < resultHeight && currentCol < resultWidth)
            {
                double tmpSum = 0.0f;

                for (int k = 0; k < leftWidth; ++k)
                {
                    double leftValue = left[(currentRow)*leftWidth + k];
                    double rightValue = right[k * resultWidth + currentCol];
                    tmpSum += leftValue * rightValue;
                }
                result[(currentRow)*resultWidth + currentCol] = tmpSum;
            }

            //outer threads may need to do extra work because the matrix might not fit nicely on the grid
            if (blockIdx.y + 1 == gridDim.y && totalGridHeight < resultHeight)
            {
                int extraRow = currentRow + blockDim.y;

                if (extraRow < resultHeight && currentCol < resultWidth)
                {
                    double tmpSum = 0.0f;

                    for (int k = 0; k < leftWidth; ++k)
                    {
                        double leftValue = left[(extraRow)*leftWidth + k];
                        double rightValue = right[k * resultWidth + currentCol];
                        tmpSum += leftValue * rightValue;
                    }
                    result[(extraRow)*resultWidth + currentCol] = tmpSum;
                }
            }

            if (blockIdx.x + 1 == gridDim.x && totalGridWidth < resultWidth)
            {
                int extraCol = currentCol + blockDim.x;

                if (currentRow < resultHeight && extraCol < resultWidth)
                {
                    double tmpSum = 0.0f;

                    for (int k = 0; k < leftWidth; ++k)
                    {
                        double leftValue = left[(currentRow)*leftWidth + k];
                        double rightValue = right[k * resultWidth + extraCol];
                        tmpSum += leftValue * rightValue;
                    }
                    result[(currentRow)*resultWidth + extraCol] = tmpSum;
                }
            }

            if (blockIdx.x + 1 == gridDim.x && totalGridWidth < resultWidth && blockIdx.y + 1 == gridDim.y && totalGridHeight < resultHeight)
            {
                int extraCol = currentCol + blockDim.x;
                int extraRow = currentRow + blockDim.y;
                if (extraRow < resultHeight && extraCol < resultWidth)
                {
                    float tmpSum = 0.0f;

                    for (int k = 0; k < leftWidth; ++k)
                    {
                        double leftValue = left[(extraRow)*leftWidth + k];
                        double rightValue = right[k * resultWidth + extraCol];
                        tmpSum += leftValue * rightValue;
                    }
                    result[(extraRow)*resultWidth + extraCol] = tmpSum;
                }
            }
        }
    }
}

hipError_t launchMultiplyKernel(dim3 blocksPerGrid, dim3 threadsPerBlock, int valueColsPerThread, int valueRowsPerThread, int resultWidth, int resultHeight, int leftWidth, double* dev_left, double* dev_right, double* dev_result)
{
    multiplyKernel << <blocksPerGrid, threadsPerBlock >> > (resultWidth, resultHeight, leftWidth, valueColsPerThread, valueRowsPerThread, dev_left, dev_right, dev_result);
    return hipGetLastError();
}

